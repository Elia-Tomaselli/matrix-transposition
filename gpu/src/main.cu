#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include "matrix.cuh"
#include "cuda_error.cuh"

#define LOOPS 5

#ifndef TILE_SIZE
#define TILE_SIZE 32 // Default value if not defined
#endif

#define LOG2(x) (                               \
    (x) <= 0 ? -1 : ((x) & ((x) - 1)) != 0 ? -1 \
                                           : __builtin_ctz(x))

__global__ void matrix_transpose_naive(matrix_type *dst, matrix_type *src, uint64_t size)
{
  int tilesPerRow = size / TILE_SIZE;

  int tileX = blockIdx.x % tilesPerRow;
  int tileY = blockIdx.x / tilesPerRow;

  int x = tileX * TILE_SIZE + (threadIdx.x % TILE_SIZE);
  int y = tileY * TILE_SIZE + (threadIdx.x / TILE_SIZE);

  dst[x * size + y] = src[y * size + x];
}

__global__ void matrix_transpose_optimized(matrix_type *dst, matrix_type *src, uint64_t size)
{
  // Shared memory for the tile, the extra element is for bank conflict avoidance
  __shared__ matrix_type tile[TILE_SIZE][TILE_SIZE + 1];

  int tilesPerRow = size / TILE_SIZE;

  int tileX = blockIdx.x % tilesPerRow;
  int tileY = blockIdx.x / tilesPerRow;

  // Local coordinates are the coordinates of the element within the block
  // Global coordinates are the coordinates of the element within the whole matrix
  int localX = threadIdx.x % TILE_SIZE;
  int localY = threadIdx.x / TILE_SIZE;
  int globalX = tileX * TILE_SIZE + localX;
  int globalY = tileY * TILE_SIZE + localY;

  tile[localY][localX] = src[globalY * size + globalX];

  __syncthreads();

  dst[globalY * size + globalX] = tile[localX][localY];
}

inline uint8_t parse_arguments(int argc, char **argv)
{
  if (argc != 2)
  {
    fprintf(stderr, "Usage: %s <exponent>\n", argv[0]);
    exit(1);
  }

  uint8_t exponent = (uint8_t)atoi(argv[1]);
  if (exponent == 0 && argv[1][0] != '0')
  {
    fprintf(stderr, "Error: Invalid input\n");
    exit(1);
  }

  uint8_t min_exponent = LOG2(TILE_SIZE);
  if (exponent < min_exponent)
  {
    fprintf(stderr, "Error: Exponent must be greater than or equal to %d\n", min_exponent);
    exit(1);
  }

  return exponent;
}

int main(int argc, char **argv)
{
  uint8_t exponent = parse_arguments(argc, argv);
  uint64_t size = 1 << exponent;

  matrix_t matrix;
  matrix_alloc(&matrix, size);
  matrix_t matrixT;
  matrix_alloc(&matrixT, size);

  srand(time(NULL));
  matrix_init_random(&matrix, size);

  // Get max threads per block
  int device;
  gpuErrchk(hipGetDevice(&device));

  int maxThreadsPerBlock;
  gpuErrchk(hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device));
  // printf("Max threads per block: %d\n", maxThreadsPerBlock);

  int sharedMemPerBlock;
  gpuErrchk(hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device));
  // printf("Max shared memory per block: %d\n", sharedMemPerBlock);

  int block_size = TILE_SIZE * TILE_SIZE;
  int grid_size = size * size / block_size;
  // printf("Block size: %d\tGrid size: %d\n", block_size, grid_size);

  hipEvent_t start;
  gpuErrchk(hipEventCreate(&start));
  hipEvent_t stop;
  gpuErrchk(hipEventCreate(&stop));

#ifdef NAIVE
  printf("Naive\n");

  // Warmup kernel
  matrix_transpose_naive<<<grid_size, block_size>>>(matrixT.d, matrix.d, size);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < LOOPS; i++)
  {
    gpuErrchk(hipEventRecord(start));

    matrix_transpose_naive<<<grid_size, block_size>>>(matrixT.d, matrix.d, size);

    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipPeekAtLastError());

    float duration;
    gpuErrchk(hipEventElapsedTime(&duration, start, stop));
    printf("Time: %f ms\n", duration);
  }

#elif OPTIMIZED
  printf("Optimized\n");
  
  // Warmup kernel
  matrix_transpose_optimized<<<grid_size, block_size>>>(matrixT.d, matrix.d, size);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < LOOPS; i++)
  {
    gpuErrchk(hipEventRecord(start));

    matrix_transpose_optimized<<<grid_size, block_size>>>(matrixT.d, matrix.d, size);

    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipPeekAtLastError());

    float duration;
    gpuErrchk(hipEventElapsedTime(&duration, start, stop));
    printf("Time: %f ms\n", duration);
  }

#endif

  gpuErrchk(hipMemcpy(matrixT.h, matrixT.d, size * size * sizeof(matrix_type), hipMemcpyDeviceToHost));

  // matrix_print(&matrix, size);
  // matrix_print(&matrixT, size);

  matrix_free(&matrix);
  matrix_free(&matrixT);

  gpuErrchk(hipDeviceReset());

  return 0;
}
