#include "matrix.cuh"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_error.cuh"

void matrix_alloc(matrix_t *matrix, uint64_t size)
{
  matrix_type *h_matrix;
  gpuErrchk(hipHostMalloc((void **)&h_matrix, size * size * sizeof(matrix_type)));
  matrix->h = h_matrix;

  matrix_type *d_matrix;
  gpuErrchk(hipMalloc((void **)&d_matrix, size * size * sizeof(matrix_type)));
  matrix->d = d_matrix;
}

void matrix_free(matrix_t *matrix)
{
  gpuErrchk(hipHostFree(matrix->h));
  gpuErrchk(hipFree(matrix->d));
}

void matrix_init_random(matrix_t *matrix, uint64_t size)
{
  for (uint64_t i = 0; i < size * size; i++)
    matrix->h[i] = (matrix_type)rand() / RAND_MAX;

  gpuErrchk(hipMemcpy(matrix->d, matrix->h, size * size * sizeof(matrix_type), hipMemcpyHostToDevice));
}

void matrix_print(matrix_t* matrix, uint64_t size)
{
  printf("[\n");
  for (uint64_t i = 0; i < size; i++)
  {
    printf("[");
    for (uint64_t j = 0; j < size; j++)
    {
      uint64_t index = i * size + j;
      printf(MATRIX_TYPE_FORMAT_SPECIFIER, matrix->h[index]);
      if (j != size - 1)
        printf(", ");
    }
    printf("]");
    if (i != size - 1)
      printf(",\n");
  }
  printf("\n]\n");
}
